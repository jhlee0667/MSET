#include "hip/hip_runtime.h"
#include "mset.cuh"

// constructor
mset::paras::paras(std::array<size_t,3> rec_size) : rec_size(rec_size){

    RVol = new float[rec_size[0]*rec_size[1]*rec_size[2]];
    init_wave2D = new float2[rec_size[1]*rec_size[2]];
    prop2D = new float2[rec_size[1]*rec_size[2]];
    ifftshift_prop2D = new float2[rec_size[1]*rec_size[2]];
    error_array = new float[rec_size[1]*rec_size[2]];

    hipMalloc(&dev_RVol, sizeof(float) * rec_size[0]*rec_size[1]*rec_size[2]);
    hipMalloc(&dev_trans_fun, sizeof(float2) * rec_size[0]*rec_size[1]*rec_size[2]);
    hipMalloc(&dev_conj_trans_fun, sizeof(float2) * rec_size[0]*rec_size[1]*rec_size[2]);
    hipMalloc(&dev_ifftshift_prop2D, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_ifftshift_back_prop2D, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_init_wave2D, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_tmp_init_wave2D, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_fsfactor, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_shift_fsfactor, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_wave2D, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_save_wave3D, sizeof(float2) * rec_size[0]*rec_size[1]*rec_size[2]);
    hipMalloc(&dev_tmp_4Dcell_array, sizeof(float) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_tmp_ifftshift_4Dcell_array, sizeof(float) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_fourier_inten, sizeof(float) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_residual_vector, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_grad_complex, sizeof(float2) * rec_size[0]*rec_size[1]*rec_size[2]);
    hipMalloc(&dev_grad, sizeof(float) * rec_size[0]*rec_size[1]*rec_size[2]);
    hipMalloc(&dev_grad2d, sizeof(float2) * rec_size[0]*rec_size[1]*rec_size[2]);
    hipMalloc(&dev_tmp, sizeof(float2) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_error_array, sizeof(float) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_tmp_error_array, sizeof(float) * rec_size[1]*rec_size[2]);
    hipMalloc(&dev_tmp_1D, sizeof(float)*1); 

    // init.
    hipMemset((void*) dev_error_array, 0, sizeof(float) * rec_size[1]*rec_size[2]);
}

// destructor
mset::paras::~paras(){
    //std::cout << "Call destructor" <<std::endl;

    hipFree (dev_RVol);
    hipFree (dev_trans_fun); 
    hipFree (dev_conj_trans_fun); 
    hipFree (dev_ifftshift_prop2D);
    hipFree (dev_ifftshift_back_prop2D);
    hipFree (dev_init_wave2D);
    hipFree (dev_tmp_init_wave2D);
    hipFree (dev_fsfactor);
    hipFree (dev_shift_fsfactor);
    hipFree (dev_wave2D);
    hipFree (dev_save_wave3D);
    hipFree (dev_tmp_4Dcell_array);
    hipFree (dev_tmp_ifftshift_4Dcell_array);
    hipFree (dev_fourier_inten);
    hipFree (dev_residual_vector);
    hipFree (dev_grad_complex);
    hipFree (dev_grad);
    hipFree (dev_grad2d);
    hipFree (dev_tmp);
    hipFree (dev_error_array);
    hipFree (dev_tmp_error_array);
    hipFree (dev_tmp_1D);
 
    delete [] RVol;
    delete [] init_wave2D;
    delete [] prop2D;  
    delete [] ifftshift_prop2D;
    delete [] error_array;
}






void mset::v0::run_main_mset(std::shared_ptr<mset::paras> stem_paras){

    unsigned long dims[3] = {stem_paras->rec_size[0], stem_paras->rec_size[1], stem_paras->rec_size[2]};
    size_t pot_array_centers[2] = {(size_t) round((stem_paras->rec_size[1]+1.0f)/2), (size_t) round((stem_paras->rec_size[1]+1.0f)/2)};

    // --------------------------------------------------------
    const int BATCH = 1;
    int dims_2d[2] = {(int) dims[1], (int) dims[2]};
    hipfftComplex ifft_scale_factor = make_hipFloatComplex ((float) 1/((float) dims[1]* (float) dims[2]), 0.0f);
    hipfftComplex complex_sigma = make_hipFloatComplex ((float) stem_paras->sigma[0], 0.0f);
    hipfftComplex complex_i = make_hipFloatComplex (0.0f, 1.0f);
    hipfftComplex complex_pot_size = make_hipFloatComplex ((float) stem_paras->potential_pixelsize[0], 0.0f);
    float tmp_1D;
    //std::cout<< "(" << complex_sigma.x << ", "<< complex_sigma.y << ")" << std::endl; 

    // Create a 2D FFT plan. 
    hipfftHandle plan;
    if (hipfftPlanMany(&plan, 2, (int *) dims_2d,
                        NULL, 1, 0, // *inembed, istride, idist 
                        NULL, 1, 0, // *onembed, ostride, odist
                        HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT error: Plan creation failed");
        return;	
    }	
    // --------------------------------------------------------

    /* -------------------  main part  --------------------- */
    for (size_t scan_p = 0; scan_p < stem_paras->num_scan_pos[0]; ++scan_p){
   
        // ------------ init. wave2D ------------------------------
        hipMemcpy(stem_paras->dev_wave2D, stem_paras->dev_init_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        
        // shift init wave using Fourier shift thm
        // CUFFT INVERSE
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
            return;	
        }
        // scaling
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

        fourier_shift_factor_dev<<<256, 256>>>(stem_paras->dev_fsfactor, (int) dims[1], (int) dims[2], (float) stem_paras->scan_ylist[scan_p] - (float) pot_array_centers[0], (float) stem_paras->scan_xlist[scan_p] - (float) pot_array_centers[1]);
        circular_shift2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_fsfactor, (hipfftComplex *) stem_paras->dev_shift_fsfactor, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_shift_fsfactor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
        
        // CUFFT FORWARD
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }
        hipMemcpy(stem_paras->dev_tmp_init_wave2D, stem_paras->dev_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 
        // --------------------------------------------------------
                    
        // ------------ transmision function ----------------------
        // copy RVol -> trans
        datatransfer_F2C<<<256, 256>>>(stem_paras->dev_trans_fun, stem_paras->dev_RVol, (size_t) dims[0]*dims[1]*dims[2]);


        // calculate trans function
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_pot_size, (hipfftComplex *) stem_paras->dev_trans_fun, dims[0]*dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_sigma, (hipfftComplex *) stem_paras->dev_trans_fun, dims[0]*dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_i, (hipfftComplex *) stem_paras->dev_trans_fun, dims[0]*dims[1]*dims[2]);
        
        // slice_binning
        for (size_t i = 0; i < dims[0]; ++i){
            if (i % (size_t) stem_paras->slice_binning[0] != 0){
                CuPointwiseAdd<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun + i*dims[1]*dims[2], (hipfftComplex *) stem_paras->dev_trans_fun + (i / (size_t) stem_paras->slice_binning[0])* (size_t) stem_paras->slice_binning[0] *dims[1]*dims[2], dims[1]*dims[2]);
            }           
        }

        CuPointwiseExp<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun, dims[0]*dims[1]*dims[2]);
        hipMemcpy(stem_paras->dev_conj_trans_fun, stem_paras->dev_trans_fun, sizeof(float2)*dims[0]*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_conj_trans_fun, dims[0]*dims[1]*dims[2]);
        // -------------------------------------------------------- 

        
        // FORWARD PROPAGATION calculation
        for (size_t i = 0; i< dims[0]; ++i){
            if (i % (size_t) stem_paras->slice_binning[0] == 0){
                // trans_function * wave_2D
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2], i);

                // CUFFT FORWARD
                if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
                    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
                    return;	
                }

                // ifftshift_prop2D * wave_2D
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_ifftshift_prop2D, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]); 

                // CUFFT INVERSE
                if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
                    fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
                    return;	
                }
                // scaling
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
                
                // save the 2D wave funtion into 3D volume
                hipMemcpy(stem_paras->dev_save_wave3D + i*dims[1]*dims[2], stem_paras->dev_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 
            }
        }
        // -------------------------------------------------------- 
        
        // calculate residual vector
        // CUFFT FORWARD            
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }

        CuPointwiseAbsSquare<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]);

        CuPointwiseAdd<<<1024, 256>>>((float) pow(10,-30), (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]); // preventing divergence 
        // square root
        CuPointwisePow<<<1024, 256>>>((float *) stem_paras->dev_fourier_inten, 0.5f, dims[1]*dims[2]);
        

        // get measured 4D STEM data
        stem_paras->pMxCell_4DSTEM_element = mxGetCell(stem_paras->pMxCell_4DSTEM, scan_p);
        stem_paras->tmp_4Dcell_array = mxGetSingles(stem_paras->pMxCell_4DSTEM_element);

        
        // ifftshift 4D STEM data
        hipMemcpy(stem_paras->dev_tmp_4Dcell_array, stem_paras->tmp_4Dcell_array, sizeof(float)*dims[1]*dims[2], hipMemcpyHostToDevice);
        
        circular_shift2D_dev<<<256, 256>>>((float *) stem_paras->dev_tmp_4Dcell_array, (float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        // square root
        CuPointwisePow<<<1024, 256>>>((float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, 0.5f, dims[1]*dims[2]);
        

        // calculate error
        hipMemcpy(stem_paras->dev_tmp_error_array, stem_paras->dev_tmp_ifftshift_4Dcell_array, sizeof(float)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 
        CuPointwiseSub<<<1024, 256>>>((float *) stem_paras->dev_fourier_inten, (float *) stem_paras->dev_tmp_error_array, dims[1]*dims[2]); 
        CuPointwiseAbsSquare<<<1024, 256>>>((float *) stem_paras->dev_tmp_error_array, dims[1]*dims[2]);
        CuPointwiseAdd<<<1024, 256>>>((float *) stem_paras->dev_tmp_error_array, (float *) stem_paras->dev_error_array, dims[1]*dims[2]);

        // divide estimated Fourier intensity by measured Fourier intensity (result: second argument)
        CuPointwiseDiv<<<1024, 256>>>((float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]); 
        

        // copy the above result (dev_fourier_inten) -> dev_residual_vector
        datatransfer_F2C<<<256, 256>>>(stem_paras->dev_residual_vector, stem_paras->dev_fourier_inten, (size_t) dims[1]*dims[2]);
        CuPointwiseSub<<<1024, 256>>>(make_hipFloatComplex (1.0f, 0.0f), stem_paras->dev_residual_vector, (size_t) dims[1]*dims[2]);
            
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_residual_vector, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

        // -------------------------------------------------------- 

        
        // BACK PROPAGATION calculation
        for (int i = dims[0]-1; i >= 0; --i){
            if (i % (size_t) stem_paras->slice_binning[0] == 0){
                // ifftshift_back_prop2D * wave_2D 
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_ifftshift_back_prop2D, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

                // CUFFT INVERSE
                if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
                    fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
                    return;	
                }
                // scaling
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

                // save 3D grad
                hipMemcpy(stem_paras->dev_grad2d, stem_paras->dev_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
                hipMemcpy(stem_paras->dev_tmp, stem_paras->dev_save_wave3D + i*dims[1]*dims[2], sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
                CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_tmp, (hipfftComplex *) stem_paras->dev_grad2d, dims[1]*dims[2]); 
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_conj_trans_fun, (hipfftComplex *) stem_paras->dev_grad2d, dims[1]*dims[2], i);
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_i, (hipfftComplex *) stem_paras->dev_grad2d, dims[1]*dims[2]);
                hipMemcpy(stem_paras->dev_grad_complex + i*dims[1]*dims[2], stem_paras->dev_grad2d, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 

                // conjugate_trans_function * wave_2D
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_conj_trans_fun, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2], i);

                // CUFFT FORWARD
                if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
                    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
                    return;	
                }
            }
        }
        // CUFFT INVERSE
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
            return;	
        }
        // scaling
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

        // slice_binning 
        for (size_t i = 0; i < dims[0]; ++i){
            if (i % (size_t) stem_paras->slice_binning[0] != 0){
                hipMemcpy(stem_paras->dev_grad_complex + i*dims[1]*dims[2], stem_paras->dev_grad_complex + (i / (size_t) stem_paras->slice_binning[0])* (size_t) stem_paras->slice_binning[0]*dims[1]*dims[2],  sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 
            }           
        }


        // update object
        datatransfer_C2F<<<256, 256>>>(stem_paras->dev_grad, stem_paras->dev_grad_complex, (size_t) dims[0]*dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((float) stem_paras->step_size[0], (float *) stem_paras->dev_grad, dims[0]*dims[1]*dims[2]); 
        CuPointwiseAdd<<<1024, 256>>>((float *) stem_paras->dev_grad, (float *) stem_paras->dev_RVol, dims[0]*dims[1]*dims[2]); 
        // --------------------------------------------------------


        // update probe wave
        hipMemcpy(stem_paras->dev_tmp, stem_paras->dev_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);

        // shift back wave using Fourier shift thm
        // CUFFT INVERSE
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_tmp, (hipfftComplex *) stem_paras->dev_tmp, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
            return;	
        }
        // scaling
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);

        fourier_shift_factor_dev<<<256, 256>>>(stem_paras->dev_fsfactor, (int) dims[1], (int) dims[2], -1.0f* (float) stem_paras->scan_ylist[scan_p] + (float) pot_array_centers[0], -1.0f* (float) stem_paras->scan_xlist[scan_p] + (float) pot_array_centers[1]);
        circular_shift2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_fsfactor, (hipfftComplex *) stem_paras->dev_shift_fsfactor, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_shift_fsfactor, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        
        // CUFFT FORWARD
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_tmp, (hipfftComplex *) stem_paras->dev_tmp, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }

        CuPointwiseMul<<<1024, 256>>>(make_hipFloatComplex (-1.0f * (float) stem_paras->step_size[1], 0.0f), (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        CuPointwiseAdd<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_tmp, (hipfftComplex *) stem_paras->dev_init_wave2D, dims[1]*dims[2]);  
        // --------------------------------------------------------


        // update scan position
        CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_tmp_init_wave2D, dims[1]*dims[2]);
        // x update
        gradient_2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_tmp_init_wave2D, (hipfftComplex *) stem_paras->dev_tmp, (int) dims[1], (int) dims[2], (int) 2);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        tmp_1D = Cusum_real((hipfftComplex *) stem_paras->dev_tmp, (size_t) dims[1]*dims[2]);
        stem_paras->scan_xlist[scan_p] += (float) stem_paras->step_size[2] * (float) tmp_1D; 
        // y update
        gradient_2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_tmp_init_wave2D, (hipfftComplex *) stem_paras->dev_tmp, (int) dims[1], (int) dims[2], (int) 1);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        tmp_1D = Cusum_real((hipfftComplex *) stem_paras->dev_tmp, (size_t) dims[1]*dims[2]);
        stem_paras->scan_ylist[scan_p] += (float) stem_paras->step_size[2] * (float) tmp_1D;
        // --------------------------------------------------------
        
        
        // positivity
        //CuPositivity<<<256, 256>>>((float *) stem_paras->dev_RVol, dims[0]*dims[1]*dims[2]);
    }
}










void mset::v0::run_main_sset(std::shared_ptr<mset::paras> stem_paras){

    unsigned long dims[3] = {stem_paras->rec_size[0], stem_paras->rec_size[1], stem_paras->rec_size[2]};
    size_t pot_array_centers[2] = {(size_t) round((stem_paras->rec_size[1]+1.0f)/2), (size_t) round((stem_paras->rec_size[1]+1.0f)/2)};

    // --------------------------------------------------------
    const int BATCH = 1;
    int dims_2d[2] = {(int) dims[1], (int) dims[2]};
    hipfftComplex ifft_scale_factor = make_hipFloatComplex ((float) 1/((float) dims[1]* (float) dims[2]), 0.0f);
    hipfftComplex complex_sigma = make_hipFloatComplex ((float) stem_paras->sigma[0], 0.0f);
    hipfftComplex complex_i = make_hipFloatComplex (0.0f, 1.0f);
    hipfftComplex complex_pot_size = make_hipFloatComplex ((float) stem_paras->potential_pixelsize[0], 0.0f); 
    float tmp_1D;
    //std::cout<< "(" << complex_sigma.x << ", "<< complex_sigma.y << ")" << std::endl; 

    // Create a 2D FFT plan. 
    hipfftHandle plan;
    if (hipfftPlanMany(&plan, 2, (int *) dims_2d,
                        NULL, 1, 0, // *inembed, istride, idist 
                        NULL, 1, 0, // *onembed, ostride, odist
                        HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT error: Plan creation failed");
        return;	
    }	
    // --------------------------------------------------------

    /* -------------------  main part  --------------------- */
    for (size_t scan_p = 0; scan_p < stem_paras->num_scan_pos[0]; ++scan_p){
   
        // ------------ init. wave2D ------------------------------
        hipMemcpy(stem_paras->dev_wave2D, stem_paras->dev_init_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        
        // shift init wave using Fourier shift thm
        // CUFFT INVERSE
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
            return;	
        }
        // scaling
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

        fourier_shift_factor_dev<<<256, 256>>>(stem_paras->dev_fsfactor, (int) dims[1], (int) dims[2], (float) stem_paras->scan_ylist[scan_p] - (float) pot_array_centers[0], (float) stem_paras->scan_xlist[scan_p] - (float) pot_array_centers[1]);
        circular_shift2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_fsfactor, (hipfftComplex *) stem_paras->dev_shift_fsfactor, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_shift_fsfactor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
        
        // CUFFT FORWARD
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }
        hipMemcpy(stem_paras->dev_tmp_init_wave2D, stem_paras->dev_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        // -------------------------------------------------------- 
                     

        // ------------ transmision function ----------------------
        // copy RVol -> trans
        datatransfer_F2C<<<256, 256>>>(stem_paras->dev_trans_fun, stem_paras->dev_RVol, (size_t) dims[0]*dims[1]*dims[2]);


        // calculate trans function
        for (size_t i = 0+1; i< dims[0]; ++i){
            CuPointwiseAdd<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun+ i*dims[1]*dims[2],(hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);  
        }
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_pot_size, (hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_sigma, (hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_i, (hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);
        CuPointwiseExp<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);
        hipMemcpy(stem_paras->dev_conj_trans_fun, stem_paras->dev_trans_fun, sizeof(float2)*dims[0]*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_conj_trans_fun, dims[1]*dims[2]);
        // -------------------------------------------------------- 

        
        // FORWARD PROPAGATION calculation
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
        // -------------------------------------------------------- 
        
        // calculate residual vector
        // CUFFT FORWARD            
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }

        CuPointwiseAbsSquare<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]);

        CuPointwiseAdd<<<1024, 256>>>((float) pow(10,-30), (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]); // preventing divergence
        // square root
        CuPointwisePow<<<1024, 256>>>((float *) stem_paras->dev_fourier_inten, 0.5f, dims[1]*dims[2]);
        

        // get measured 4D STEM data
        stem_paras->pMxCell_4DSTEM_element = mxGetCell(stem_paras->pMxCell_4DSTEM, scan_p); //(scan_y-1)*scan_size[1]+(scan_x-1));
        stem_paras->tmp_4Dcell_array = mxGetSingles(stem_paras->pMxCell_4DSTEM_element);

        
        // ifftshift 4D STEM data
        hipMemcpy(stem_paras->dev_tmp_4Dcell_array, stem_paras->tmp_4Dcell_array, sizeof(float)*dims[1]*dims[2], hipMemcpyHostToDevice);
        
        circular_shift2D_dev<<<256, 256>>>((float *) stem_paras->dev_tmp_4Dcell_array, (float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        // square root
        CuPointwisePow<<<1024, 256>>>((float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, 0.5f, dims[1]*dims[2]);
        

        // calculate error
        hipMemcpy(stem_paras->dev_tmp_error_array, stem_paras->dev_tmp_ifftshift_4Dcell_array, sizeof(float)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 
        CuPointwiseSub<<<1024, 256>>>((float *) stem_paras->dev_fourier_inten, (float *) stem_paras->dev_tmp_error_array, dims[1]*dims[2]); 
        CuPointwiseAbsSquare<<<1024, 256>>>((float *) stem_paras->dev_tmp_error_array, dims[1]*dims[2]);
        CuPointwiseAdd<<<1024, 256>>>((float *) stem_paras->dev_tmp_error_array, (float *) stem_paras->dev_error_array, dims[1]*dims[2]);

        // divide estimated Fourier intensity by measured Fourier intensity (result: second argument)
        CuPointwiseDiv<<<1024, 256>>>((float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]); 
        

        // copy the above result (dev_fourier_inten) -> dev_residual_vector
        datatransfer_F2C<<<256, 256>>>(stem_paras->dev_residual_vector, stem_paras->dev_fourier_inten, (size_t) dims[1]*dims[2]);
        CuPointwiseSub<<<1024, 256>>>(make_hipFloatComplex (1.0f, 0.0f), stem_paras->dev_residual_vector, (size_t) dims[1]*dims[2]);
            
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_residual_vector, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
        // --------------------------------------------------------  

        

        // BACK PROPAGATION calculation
        // CUFFT INVERSE
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
        return;	
        }
        // scaling
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

        // circular_shift2D_dev<<<256, 256>>>(stem_paras->dev_init_wave2D, stem_paras->dev_tmp, (int) dims[1], (int) dims[2], (int) (((float) scan_y- (float) scan_array_centers[0])/po_pr_ratio), (int) (((float) scan_x- (float) scan_array_centers[1])/po_pr_ratio));
        hipMemcpy(stem_paras->dev_tmp, stem_paras->dev_tmp_init_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_conj_trans_fun, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]); 
        hipMemcpy(stem_paras->dev_grad2d, stem_paras->dev_tmp, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_i, (hipfftComplex *) stem_paras->dev_grad2d, dims[1]*dims[2]);
        
        // back projection
        for (int i = dims[0]-1; i >= 0; --i){
            hipMemcpy(stem_paras->dev_grad_complex + i*dims[1]*dims[2], stem_paras->dev_grad2d, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        } 



        // update object
        datatransfer_C2F<<<256, 256>>>(stem_paras->dev_grad, stem_paras->dev_grad_complex, (size_t) dims[0]*dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((float) stem_paras->step_size[0], (float *) stem_paras->dev_grad, dims[0]*dims[1]*dims[2]); 
        CuPointwiseAdd<<<1024, 256>>>((float *) stem_paras->dev_grad, (float *) stem_paras->dev_RVol, dims[0]*dims[1]*dims[2]); 
        // --------------------------------------------------------


        // update probe wave
        hipMemcpy(stem_paras->dev_tmp, stem_paras->dev_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);

        // shift back wave using Fourier shift thm
        // CUFFT INVERSE
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_tmp, (hipfftComplex *) stem_paras->dev_tmp, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
            return;	
        }
        // scaling
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);

        fourier_shift_factor_dev<<<256, 256>>>(stem_paras->dev_fsfactor, (int) dims[1], (int) dims[2], -1.0f* (float) stem_paras->scan_ylist[scan_p] + (float) pot_array_centers[0], -1.0f* (float) stem_paras->scan_xlist[scan_p] + (float) pot_array_centers[1]);
        circular_shift2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_fsfactor, (hipfftComplex *) stem_paras->dev_shift_fsfactor, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_shift_fsfactor, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        
        // CUFFT FORWARD
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_tmp, (hipfftComplex *) stem_paras->dev_tmp, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }

        CuPointwiseMul<<<1024, 256>>>(make_hipFloatComplex (-1.0f * (float) stem_paras->step_size[1], 0.0f), (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        CuPointwiseAdd<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_tmp, (hipfftComplex *) stem_paras->dev_init_wave2D, dims[1]*dims[2]);  
        // --------------------------------------------------------


        // update scan position
        CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_tmp_init_wave2D, dims[1]*dims[2]);
        // x update
        gradient_2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_tmp_init_wave2D, (hipfftComplex *) stem_paras->dev_tmp, (int) dims[1], (int) dims[2], (int) 2);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        tmp_1D = Cusum_real((hipfftComplex *) stem_paras->dev_tmp, (size_t) dims[1]*dims[2]);
        stem_paras->scan_xlist[scan_p] += (float) stem_paras->step_size[2] * (float) tmp_1D; 
        // y update
        gradient_2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_tmp_init_wave2D, (hipfftComplex *) stem_paras->dev_tmp, (int) dims[1], (int) dims[2], (int) 1);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_tmp, dims[1]*dims[2]);
        tmp_1D = Cusum_real((hipfftComplex *) stem_paras->dev_tmp, (size_t) dims[1]*dims[2]);
        stem_paras->scan_ylist[scan_p] += (float) stem_paras->step_size[2] * (float) tmp_1D;
        // --------------------------------------------------------


        // positivity
        //CuPositivity<<<256, 256>>>((float *) stem_paras->dev_RVol, dims[0]*dims[1]*dims[2]);
        
    }
}










void mset::v0::run_error_mset(std::shared_ptr<mset::paras> stem_paras){

    unsigned long dims[3] = {stem_paras->rec_size[0], stem_paras->rec_size[1], stem_paras->rec_size[2]};
    size_t pot_array_centers[2] = {(size_t) round((stem_paras->rec_size[1]+1.0f)/2), (size_t) round((stem_paras->rec_size[1]+1.0f)/2)};


    // --------------------------------------------------------
    const int BATCH = 1;
    int dims_2d[2] = {(int) dims[1], (int) dims[2]};
    hipfftComplex ifft_scale_factor = make_hipFloatComplex ((float) 1/((float) dims[1]* (float) dims[2]), 0.0f);
    hipfftComplex complex_sigma = make_hipFloatComplex ((float) stem_paras->sigma[0], 0.0f);
    hipfftComplex complex_i = make_hipFloatComplex (0.0f, 1.0f);
    hipfftComplex complex_pot_size = make_hipFloatComplex ((float) stem_paras->potential_pixelsize[0], 0.0f); 
    //std::cout<< "(" << complex_sigma.x << ", "<< complex_sigma.y << ")" << std::endl; 

    // Create a 2D FFT plan. 
    hipfftHandle plan;
    if (hipfftPlanMany(&plan, 2, (int *) dims_2d,
                        NULL, 1, 0, // *inembed, istride, idist 
                        NULL, 1, 0, // *onembed, ostride, odist
                        HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT error: Plan creation failed");
        return;	
    }	
    // --------------------------------------------------------

    /* -------------------  main part  --------------------- */
    for (size_t scan_p = 0; scan_p < stem_paras->num_scan_pos[0]; ++scan_p){
   
        // ------------ init. wave2D ------------------------------
        hipMemcpy(stem_paras->dev_wave2D, stem_paras->dev_init_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        
        // shift init wave using Fourier shift thm
        // CUFFT INVERSE
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
            return;	
        }
        // scaling
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

        fourier_shift_factor_dev<<<256, 256>>>(stem_paras->dev_fsfactor, (int) dims[1], (int) dims[2], (float) stem_paras->scan_ylist[scan_p] - (float) pot_array_centers[0], (float) stem_paras->scan_xlist[scan_p] - (float) pot_array_centers[1]);
        circular_shift2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_fsfactor, (hipfftComplex *) stem_paras->dev_shift_fsfactor, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_shift_fsfactor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
        
        // CUFFT FORWARD
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }
        
        // ------------ transmision function ----------------------
        // copy RVol -> trans
        datatransfer_F2C<<<256, 256>>>(stem_paras->dev_trans_fun, stem_paras->dev_RVol, (size_t) dims[0]*dims[1]*dims[2]);


        // calculate trans function
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_pot_size, (hipfftComplex *) stem_paras->dev_trans_fun, dims[0]*dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_sigma, (hipfftComplex *) stem_paras->dev_trans_fun, dims[0]*dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_i, (hipfftComplex *) stem_paras->dev_trans_fun, dims[0]*dims[1]*dims[2]);

        // slice_binning
        for (size_t i = 0; i < dims[0]; ++i){
            if (i % (size_t) stem_paras->slice_binning[0] != 0){
                CuPointwiseAdd<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun + i*dims[1]*dims[2], (hipfftComplex *) stem_paras->dev_trans_fun + (i / (size_t) stem_paras->slice_binning[0])* (size_t) stem_paras->slice_binning[0] *dims[1]*dims[2], dims[1]*dims[2]);
            }           
        }

        CuPointwiseExp<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun, dims[0]*dims[1]*dims[2]);
        hipMemcpy(stem_paras->dev_conj_trans_fun, stem_paras->dev_trans_fun, sizeof(float2)*dims[0]*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_conj_trans_fun, dims[0]*dims[1]*dims[2]);
        // -------------------------------------------------------- 

        
        // FORWARD calculation
        for (size_t i = 0; i< dims[0]; ++i){
            if (i % (size_t) stem_paras->slice_binning[0] == 0){
                // trans_function * wave_2D
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2], i);

                // CUFFT FORWARD
                if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
                    fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
                    return;	
                }

                // ifftshift_prop2D * wave_2D
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_ifftshift_prop2D, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]); 

                // CUFFT INVERSE
                if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
                    fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
                    return;	
                }
                // scaling
                CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
                
                // save the 2D wave funtion into 3D volume
                hipMemcpy(stem_paras->dev_save_wave3D + i*dims[1]*dims[2], stem_paras->dev_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 
            }
        }
        // -------------------------------------------------------- 
        
        // calculate residual vector
        // CUFFT FORWARD            
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }

        CuPointwiseAbsSquare<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]);

        CuPointwiseAdd<<<1024, 256>>>((float) pow(10,-30), (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]); 
        // square root
        CuPointwisePow<<<1024, 256>>>((float *) stem_paras->dev_fourier_inten, 0.5f, dims[1]*dims[2]);
        

        // get measured 4D STEM data
        stem_paras->pMxCell_4DSTEM_element = mxGetCell(stem_paras->pMxCell_4DSTEM, scan_p);
        stem_paras->tmp_4Dcell_array = mxGetSingles(stem_paras->pMxCell_4DSTEM_element);

        
        // ifftshift 4D STEM data
        hipMemcpy(stem_paras->dev_tmp_4Dcell_array, stem_paras->tmp_4Dcell_array, sizeof(float)*dims[1]*dims[2], hipMemcpyHostToDevice);
        
        circular_shift2D_dev<<<256, 256>>>((float *) stem_paras->dev_tmp_4Dcell_array, (float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        // square root
        CuPointwisePow<<<1024, 256>>>((float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, 0.5f, dims[1]*dims[2]);
        

        // calculate error
        hipMemcpy(stem_paras->dev_tmp_error_array, stem_paras->dev_tmp_ifftshift_4Dcell_array, sizeof(float)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 
        CuPointwiseSub<<<1024, 256>>>((float *) stem_paras->dev_fourier_inten, (float *) stem_paras->dev_tmp_error_array, dims[1]*dims[2]); 
        CuPointwiseAbsSquare<<<1024, 256>>>((float *) stem_paras->dev_tmp_error_array, dims[1]*dims[2]);
        CuPointwiseAdd<<<1024, 256>>>((float *) stem_paras->dev_tmp_error_array, (float *) stem_paras->dev_error_array, dims[1]*dims[2]);
        // --------------------------------------------------------  

    }
}



void mset::v0::run_error_sset(std::shared_ptr<mset::paras> stem_paras){

    unsigned long dims[3] = {stem_paras->rec_size[0], stem_paras->rec_size[1], stem_paras->rec_size[2]};
    size_t pot_array_centers[2] = {(size_t) round((stem_paras->rec_size[1]+1.0f)/2), (size_t) round((stem_paras->rec_size[1]+1.0f)/2)};

    // --------------------------------------------------------
    const int BATCH = 1;
    int dims_2d[2] = {(int) dims[1], (int) dims[2]};
    hipfftComplex ifft_scale_factor = make_hipFloatComplex ((float) 1/((float) dims[1]* (float) dims[2]), 0.0f);
    hipfftComplex complex_sigma = make_hipFloatComplex ((float) stem_paras->sigma[0], 0.0f);
    hipfftComplex complex_i = make_hipFloatComplex (0.0f, 1.0f);
    hipfftComplex complex_pot_size = make_hipFloatComplex ((float) stem_paras->potential_pixelsize[0], 0.0f); 
    //std::cout<< "(" << complex_sigma.x << ", "<< complex_sigma.y << ")" << std::endl; 

    // Create a 2D FFT plan. 
    hipfftHandle plan;
    if (hipfftPlanMany(&plan, 2, (int *) dims_2d,
                        NULL, 1, 0, // *inembed, istride, idist 
                        NULL, 1, 0, // *onembed, ostride, odist
                        HIPFFT_C2C, BATCH) != HIPFFT_SUCCESS){
        fprintf(stderr, "CUFFT error: Plan creation failed");
        return;	
    }	
    // --------------------------------------------------------

    /* -------------------  main part  --------------------- */
    for (size_t scan_p = 0; scan_p < stem_paras->num_scan_pos[0]; ++scan_p){
   
        // ------------ init. wave2D ------------------------------
        hipMemcpy(stem_paras->dev_wave2D, stem_paras->dev_init_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        
        // shift init wave using Fourier shift thm
        // CUFFT INVERSE
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_BACKWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Inverse failed");
            return;	
        }
        // scaling
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) ifft_scale_factor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);

        fourier_shift_factor_dev<<<256, 256>>>(stem_paras->dev_fsfactor, (int) dims[1], (int) dims[2], (float) stem_paras->scan_ylist[scan_p] - (float) pot_array_centers[0], (float) stem_paras->scan_xlist[scan_p] - (float) pot_array_centers[1]);
        circular_shift2D_dev<<<256, 256>>>((hipfftComplex *) stem_paras->dev_fsfactor, (hipfftComplex *) stem_paras->dev_shift_fsfactor, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_shift_fsfactor, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
        
        // CUFFT FORWARD
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }
        // -------------------------------------------------------- 
                        
        // ------------ transmision function ----------------------
        // copy RVol -> trans
        datatransfer_F2C<<<256, 256>>>(stem_paras->dev_trans_fun, stem_paras->dev_RVol, (size_t) dims[0]*dims[1]*dims[2]);


        // calculate trans function
        for (size_t i = 0+1; i< dims[0]; ++i){
            CuPointwiseAdd<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun+ i*dims[1]*dims[2],(hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);  
        }
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_pot_size, (hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_sigma, (hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex) complex_i, (hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);
        CuPointwiseExp<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun, dims[1]*dims[2]);
        hipMemcpy(stem_paras->dev_conj_trans_fun, stem_paras->dev_trans_fun, sizeof(float2)*dims[0]*dims[1]*dims[2], hipMemcpyDeviceToDevice);
        CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_conj_trans_fun, dims[1]*dims[2]);
        // -------------------------------------------------------- 

        
        // FORWARD calculation
        CuPointwiseMul<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_trans_fun, (hipfftComplex *) stem_paras->dev_wave2D, dims[1]*dims[2]);
        // -------------------------------------------------------- 
        
        // calculate residual vector
        // CUFFT FORWARD            
        if (hipfftExecC2C(plan, (hipfftComplex *) stem_paras->dev_wave2D, (hipfftComplex *) stem_paras->dev_wave2D, HIPFFT_FORWARD) != HIPFFT_SUCCESS){
            fprintf(stderr, "CUFFT error: ExecC2C Forward failed");
            return;	
        }

        CuPointwiseAbsSquare<<<1024, 256>>>((hipfftComplex *) stem_paras->dev_wave2D, (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]);

        CuPointwiseAdd<<<1024, 256>>>((float) pow(10,-30), (float *) stem_paras->dev_fourier_inten, dims[1]*dims[2]); 
        // square root
        CuPointwisePow<<<1024, 256>>>((float *) stem_paras->dev_fourier_inten, 0.5f, dims[1]*dims[2]);
        

        // get measured 4D STEM data
        stem_paras->pMxCell_4DSTEM_element = mxGetCell(stem_paras->pMxCell_4DSTEM, scan_p); //(scan_y-1)*scan_size[1]+(scan_x-1));
        stem_paras->tmp_4Dcell_array = mxGetSingles(stem_paras->pMxCell_4DSTEM_element);

        
        // ifftshift 4D STEM data
        hipMemcpy(stem_paras->dev_tmp_4Dcell_array, stem_paras->tmp_4Dcell_array, sizeof(float)*dims[1]*dims[2], hipMemcpyHostToDevice);
        
        circular_shift2D_dev<<<256, 256>>>((float *) stem_paras->dev_tmp_4Dcell_array, (float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, (int) dims[1], (int) dims[2], (int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
        // square root
        CuPointwisePow<<<1024, 256>>>((float *) stem_paras->dev_tmp_ifftshift_4Dcell_array, 0.5f, dims[1]*dims[2]);
        

        // calculate error
        hipMemcpy(stem_paras->dev_tmp_error_array, stem_paras->dev_tmp_ifftshift_4Dcell_array, sizeof(float)*dims[1]*dims[2], hipMemcpyDeviceToDevice); 
        CuPointwiseSub<<<1024, 256>>>((float *) stem_paras->dev_fourier_inten, (float *) stem_paras->dev_tmp_error_array, dims[1]*dims[2]); 
        CuPointwiseAbsSquare<<<1024, 256>>>((float *) stem_paras->dev_tmp_error_array, dims[1]*dims[2]);
        CuPointwiseAdd<<<1024, 256>>>((float *) stem_paras->dev_tmp_error_array, (float *) stem_paras->dev_error_array, dims[1]*dims[2]); 
        // --------------------------------------------------------  

    }

}


