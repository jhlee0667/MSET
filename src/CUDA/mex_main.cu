#include "hip/hip_runtime.h"
#include <iostream>
#include <atomic>
#include <vector>
#include <map>
#include <cmath>
#include "mex.h"

#include "mex_tools.hpp"
#include "mset.cuh"

using namespace std;

static std::shared_ptr<mset::paras> stem_paras;

void upload(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]);
void run(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], int opti);
void error(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], int opti);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){

    std::string Run_mode = "";
    Run_mode = get_MxString(prhs[0]);

	// Choose mode
	if (Run_mode == std::string("upload")) { 
		upload(nlhs, plhs, nrhs, prhs);}
    else if (Run_mode == std::string("run_mset")){	
		run(nlhs, plhs, nrhs, prhs, 1);}
    else if (Run_mode == std::string("run_sset")){	
        run(nlhs, plhs, nrhs, prhs, 2);}
    else if (Run_mode == std::string("error_mset")){	
        error(nlhs, plhs, nrhs, prhs, 1);}
    else if (Run_mode == std::string("error_sset")){	
        error(nlhs, plhs, nrhs, prhs, 2);}
    else {
        mexErrMsgTxt("The running mode is not correct (choose 'upload' or 'run_mset' or 'run_sset' or 'error_mset' or 'error_sset')");}

    //std::cout<< Run_mode << std::endl;
}


void upload(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){

    std::vector<const mxArray*> STEM_data(prhs+1, prhs + nrhs);

    /* get a struct data (second argument) */
    // --------------------------------------------------------
    std::map<std::string, const mxArray*> map_input;
    MxStruct2vector(STEM_data[0], map_input);

    // get dimensions of init. 3D volume
    std::array<size_t,3> dims;
    get_Struct_arraysizeInfo(map_input,"RVol", dims);
    
    // create class 
    stem_paras = std::make_shared<mset::paras>(dims);


    // get Mxdata (array)
    get_Mxdata(map_input, "RVol", stem_paras->RVol, dims[0]*dims[1]*dims[2]);
    get_Mxdata(map_input, "init_wave2D", stem_paras->init_wave2D, dims[1]*dims[2]); 
    get_Mxdata(map_input, "prop2D", stem_paras->prop2D, dims[1]*dims[2]); 

    // get Mxdata (parameters)
    get_Mxdata(map_input, "alpha", stem_paras->alpha); 
    get_Mxdata(map_input, "sigma", stem_paras->sigma);
    //get_Mxdata(map_input, "inner_angle", stem_paras->inner_angle);
    //get_Mxdata(map_input, "outer_angle", stem_paras->outer_angle);
    get_Mxdata(map_input, "E0", stem_paras->E0);
    get_Mxdata(map_input, "probe_step_size", stem_paras->probe_step_size);
    get_Mxdata(map_input, "potential_pixelsize", stem_paras->potential_pixelsize);
    get_Mxdata(map_input, "N_scan_x", stem_paras->N_scan_x);
    get_Mxdata(map_input, "N_scan_y", stem_paras->N_scan_y);
    get_Mxdata(map_input, "scan_xlist", stem_paras->scan_xlist);
    get_Mxdata(map_input, "scan_ylist", stem_paras->scan_ylist);
    get_Mxdata(map_input, "num_scan_pos", stem_paras->num_scan_pos);
    get_Mxdata(map_input, "step_size", stem_paras->step_size);

    // send CPU data to GPU data
    hipMemcpy(stem_paras->dev_RVol, stem_paras->RVol, sizeof(float)*dims[0]*dims[1]*dims[2], hipMemcpyHostToDevice);
    hipMemcpy(stem_paras->dev_init_wave2D, stem_paras->init_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyHostToDevice);
    circular_shift2D((float2 *) stem_paras->prop2D, (float2 *) stem_paras->ifftshift_prop2D, (int) dims[1], (int) dims[2],(int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
    hipMemcpy(stem_paras->dev_ifftshift_prop2D, stem_paras->ifftshift_prop2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyHostToDevice);
    hipMemcpy(stem_paras->dev_ifftshift_back_prop2D, stem_paras->dev_ifftshift_prop2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
    CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_ifftshift_back_prop2D, dims[1]*dims[2]);
    // --------------------------------------------------------

    /* Get measured full 4D-STEM data */
    stem_paras->pMxCell_4DSTEM = STEM_data[1];
    stem_paras->total_num_of_4DCell = mxGetNumberOfElements(stem_paras->pMxCell_4DSTEM);
    stem_paras->cell_dims = mxGetDimensions(stem_paras->pMxCell_4DSTEM);
    // --------------------------------------------------------
   
    // Check variables for debugging
    //stem_paras->print();

}


void run(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], int opti){

    /* main calculation */
    if (opti == 1){
        mset::run_main_mset(stem_paras);}
    else if (opti == 2){
        mset::run_main_sset(stem_paras);}


    /* extract reconstructed 3D volume */
    float *output, *output_error;
    unsigned long output_dims[3] = {stem_paras->rec_size[2], stem_paras->rec_size[1], stem_paras->rec_size[0]}; 
    unsigned long output_error_dims[1] = {1}; 
    hipMemcpy(stem_paras->RVol, stem_paras->dev_RVol, sizeof(float)*output_dims[0]*output_dims[1]*output_dims[2], hipMemcpyDeviceToHost); 
    hipMemcpy(stem_paras->error_array, stem_paras->dev_error_array, sizeof(float)*output_dims[1]*output_dims[2], hipMemcpyDeviceToHost); 

    if (nlhs == 1) {
        plhs[0] = mxCreateNumericArray(3, output_dims, mxSINGLE_CLASS,  mxREAL);
        output = mxGetSingles(plhs[0]);

        for (int i = 0; i < stem_paras->rec_size[0]*stem_paras->rec_size[1]*stem_paras->rec_size[2]; ++i){ 
            output[i] = stem_paras->RVol[i];
        }
    }
    else if (nlhs == 2){
        plhs[0] = mxCreateNumericArray(3, output_dims, mxSINGLE_CLASS,  mxREAL);
        plhs[1] = mxCreateNumericArray(1, output_error_dims, mxSINGLE_CLASS,  mxREAL);
        output = mxGetSingles(plhs[0]);
        output_error = mxGetSingles(plhs[1]);

        for (int i = 0; i < stem_paras->rec_size[0]*stem_paras->rec_size[1]*stem_paras->rec_size[2]; ++i){ 
            output[i] = stem_paras->RVol[i];
        }

        output_error[0] = 0.0f;
        for (int i = 0; i < stem_paras->rec_size[1]*stem_paras->rec_size[2]; ++i){ 
            output_error[0] += stem_paras->error_array[i];
        }
        output_error[0] = output_error[0] / ((float) stem_paras->N_scan_x[0]* (float) stem_paras->N_scan_y[0])/ ((float) stem_paras->rec_size[1]* (float) stem_paras->rec_size[2]);
    }
    // ---------------------------------------------------------


}

void error(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], int opti){

    /* error calculation */
    if (opti == 1){
        mset::run_error_mset(stem_paras);}
    else if (opti == 2){
        mset::run_error_sset(stem_paras);} 

    /* extract reconstructed 3D volume */
    float *output_error;
    unsigned long output_dims[3] = {stem_paras->rec_size[2], stem_paras->rec_size[1], stem_paras->rec_size[0]}; 
    unsigned long output_error_dims[1] = {1};
    hipMemcpy(stem_paras->error_array, stem_paras->dev_error_array, sizeof(float)*output_dims[1]*output_dims[2], hipMemcpyDeviceToHost); 

  
    plhs[0] = mxCreateNumericArray(1, output_error_dims, mxSINGLE_CLASS,  mxREAL);
    output_error = mxGetSingles(plhs[0]);

    output_error[0] = 0.0f;
    for (int i = 0; i < stem_paras->rec_size[1]*stem_paras->rec_size[2]; ++i){ 
        output_error[0] += stem_paras->error_array[i];
    }
    output_error[0] = output_error[0] / ((float) stem_paras->N_scan_x[0]* (float) stem_paras->N_scan_y[0])/ ((float) stem_paras->rec_size[1]* (float) stem_paras->rec_size[2]);
    
    // ---------------------------------------------------------


}