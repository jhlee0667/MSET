#include "hip/hip_runtime.h"
#include <iostream>
#include <atomic>
#include <vector>
#include <map>
#include <cmath>
#include "mex.h"

#include "mex_tools.hpp"
#include "mset.cuh"

using namespace std;

static std::shared_ptr<mset::paras> stem_paras;

void upload(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]);
void run(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], int opti);
void error(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], int opti);

void mexFunction(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){

    std::string Run_mode = "";
    Run_mode = get_MxString(prhs[0]);

	// Choose mode
	if (Run_mode == std::string("upload")) { 
		upload(nlhs, plhs, nrhs, prhs);}
    else if (Run_mode == std::string("run_mset")){	
		run(nlhs, plhs, nrhs, prhs, 1);}
    else if (Run_mode == std::string("run_sset")){	
        run(nlhs, plhs, nrhs, prhs, 2);}
    else if (Run_mode == std::string("error_mset")){	
        error(nlhs, plhs, nrhs, prhs, 1);}
    else if (Run_mode == std::string("error_sset")){	
        error(nlhs, plhs, nrhs, prhs, 2);}
    else {
        mexErrMsgTxt("The running mode is not correct (choose 'upload' or 'run_mset' or 'run_sset' or 'error_mset' or 'error_sset')");}

    //std::cout<< Run_mode << std::endl;
}


void upload(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[]){

    std::vector<const mxArray*> STEM_data(prhs+1, prhs + nrhs);

    /* get a struct data (second argument) */
    // --------------------------------------------------------
    std::map<std::string, const mxArray*> map_input;
    MxStruct2vector(STEM_data[0], map_input);

    // get dimensions of init. 3D volume
    std::array<size_t,3> dims;
    get_Struct_arraysizeInfo(map_input,"RVol", dims);
    
    // create class 
    stem_paras = std::make_shared<mset::paras>(dims);


    // get Mxdata (array)
    get_Mxdata(map_input, "RVol", stem_paras->RVol, dims[0]*dims[1]*dims[2]);
    get_Mxdata(map_input, "init_wave2D", stem_paras->init_wave2D, dims[1]*dims[2]); 
    get_Mxdata(map_input, "prop2D", stem_paras->prop2D, dims[1]*dims[2]); 

    // get Mxdata (parameters)
    get_Mxdata(map_input, "alpha", stem_paras->alpha); 
    get_Mxdata(map_input, "sigma", stem_paras->sigma);
    get_Mxdata(map_input, "E0", stem_paras->E0);
    get_Mxdata(map_input, "potential_pixelsize", stem_paras->potential_pixelsize);
    get_Mxdata(map_input, "N_scan_x", stem_paras->N_scan_x);
    get_Mxdata(map_input, "N_scan_y", stem_paras->N_scan_y);
    get_Mxdata(map_input, "scan_xlist", stem_paras->scan_xlist);
    get_Mxdata(map_input, "scan_ylist", stem_paras->scan_ylist);
    get_Mxdata(map_input, "num_scan_pos", stem_paras->num_scan_pos);
    get_Mxdata(map_input, "step_size", stem_paras->step_size);
    get_Mxdata(map_input, "slice_binning", stem_paras->slice_binning);

    // send CPU data to GPU data
    hipMemcpy(stem_paras->dev_RVol, stem_paras->RVol, sizeof(float)*dims[0]*dims[1]*dims[2], hipMemcpyHostToDevice);
    hipMemcpy(stem_paras->dev_init_wave2D, stem_paras->init_wave2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyHostToDevice);
    circular_shift2D((float2 *) stem_paras->prop2D, (float2 *) stem_paras->ifftshift_prop2D, (int) dims[1], (int) dims[2],(int) ceil(((float)dims[1])/2.0f),(int) ceil(((float)dims[2])/2.0f));
    hipMemcpy(stem_paras->dev_ifftshift_prop2D, stem_paras->ifftshift_prop2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyHostToDevice);
    hipMemcpy(stem_paras->dev_ifftshift_back_prop2D, stem_paras->dev_ifftshift_prop2D, sizeof(float2)*dims[1]*dims[2], hipMemcpyDeviceToDevice);
    CuPointwiseConj<<<256, 256>>>((hipfftComplex *) stem_paras->dev_ifftshift_back_prop2D, dims[1]*dims[2]);
    // --------------------------------------------------------

    /* Get measured full 4D-STEM data */
    stem_paras->pMxCell_4DSTEM = STEM_data[1];
    stem_paras->total_num_of_4DCell = mxGetNumberOfElements(stem_paras->pMxCell_4DSTEM);
    stem_paras->cell_dims = mxGetDimensions(stem_paras->pMxCell_4DSTEM);
    // --------------------------------------------------------
   
    // Check variables for debugging
    //stem_paras->print();

}


void run(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], int opti){

    /* main calculation */
    if (opti == 1){
        mset::run_main_mset(stem_paras);}
    else if (opti == 2){
        mset::run_main_sset(stem_paras);}


    /* extract reconstructed 3D volume */
    float *output_rec, *output_scan_xlist, *output_scan_ylist, *output_error;
    mxComplexSingle *output_probe;
    unsigned long output_rec_dims[3] = {stem_paras->rec_size[2], stem_paras->rec_size[1], stem_paras->rec_size[0]}; 
    unsigned long output_probe_dims[2] = {stem_paras->rec_size[2], stem_paras->rec_size[1]};
    unsigned long output_scanpos_dims[1] = {(unsigned long) stem_paras->num_scan_pos[0]}; 
    unsigned long output_error_dims[1] = {1}; 
    hipMemcpy(stem_paras->RVol, stem_paras->dev_RVol, sizeof(float)*output_rec_dims[0]*output_rec_dims[1]*output_rec_dims[2], hipMemcpyDeviceToHost); 
    hipMemcpy(stem_paras->init_wave2D, stem_paras->dev_init_wave2D, sizeof(float2)*output_probe_dims[0]*output_probe_dims[1], hipMemcpyDeviceToHost); 
    hipMemcpy(stem_paras->error_array, stem_paras->dev_error_array, sizeof(float)*output_rec_dims[0]*output_rec_dims[1], hipMemcpyDeviceToHost); 
    
    // object output
    plhs[0] = mxCreateNumericArray(3, output_rec_dims, mxSINGLE_CLASS, mxREAL);
    output_rec = mxGetSingles(plhs[0]);
    for (int i = 0; i < stem_paras->rec_size[0]*stem_paras->rec_size[1]*stem_paras->rec_size[2]; ++i){ 
        output_rec[i] = stem_paras->RVol[i];
    }
    // probe wave output
    plhs[1] = mxCreateNumericArray(2, output_probe_dims, mxSINGLE_CLASS, mxCOMPLEX);
    output_probe = mxGetComplexSingles(plhs[1]);
    for (int i = 0; i < stem_paras->rec_size[1]*stem_paras->rec_size[2]; ++i){ 
        output_probe[i].real = stem_paras->init_wave2D[i].x;
        output_probe[i].imag = stem_paras->init_wave2D[i].y;
    }
    // scan position output
    plhs[2] = mxCreateNumericArray(1, output_scanpos_dims, mxSINGLE_CLASS, mxREAL);
    plhs[3] = mxCreateNumericArray(1, output_scanpos_dims, mxSINGLE_CLASS, mxREAL); 
    output_scan_xlist = mxGetSingles(plhs[2]);
    output_scan_ylist = mxGetSingles(plhs[3]);
    for (int i = 0; i < stem_paras->num_scan_pos[0]; ++i){ 
        output_scan_xlist[i] = stem_paras->scan_xlist[i];
        output_scan_ylist[i] = stem_paras->scan_ylist[i];
    }

    // error output
    if (nlhs == 5) {
        plhs[4] = mxCreateNumericArray(1, output_error_dims, mxSINGLE_CLASS, mxREAL);
        output_error = mxGetSingles(plhs[4]);

        output_error[0] = 0.0f;
        for (int i = 0; i < stem_paras->rec_size[1]*stem_paras->rec_size[2]; ++i){ 
            output_error[0] += stem_paras->error_array[i];
        }
        output_error[0] = output_error[0] / ((float) stem_paras->N_scan_x[0]* (float) stem_paras->N_scan_y[0])/ ((float) stem_paras->rec_size[1]* (float) stem_paras->rec_size[2]);
    }
    // ---------------------------------------------------------


}

void error(int nlhs, mxArray *plhs[], int nrhs, const mxArray *prhs[], int opti){

    /* error calculation */
    if (opti == 1){
        mset::run_error_mset(stem_paras);}
    else if (opti == 2){
        mset::run_error_sset(stem_paras);} 

    /* extract reconstructed 3D volume */
    float *output_error;
    unsigned long output_dims[3] = {stem_paras->rec_size[2], stem_paras->rec_size[1], stem_paras->rec_size[0]}; 
    unsigned long output_error_dims[1] = {1};
    hipMemcpy(stem_paras->error_array, stem_paras->dev_error_array, sizeof(float)*output_dims[0]*output_dims[1], hipMemcpyDeviceToHost); 

  
    plhs[0] = mxCreateNumericArray(1, output_error_dims, mxSINGLE_CLASS,  mxREAL);
    output_error = mxGetSingles(plhs[0]);

    output_error[0] = 0.0f;
    for (int i = 0; i < stem_paras->rec_size[1]*stem_paras->rec_size[2]; ++i){ 
        output_error[0] += stem_paras->error_array[i];
    }
    output_error[0] = output_error[0] / ((float) stem_paras->N_scan_x[0]* (float) stem_paras->N_scan_y[0])/ ((float) stem_paras->rec_size[1]* (float) stem_paras->rec_size[2]);
    
    // ---------------------------------------------------------


}