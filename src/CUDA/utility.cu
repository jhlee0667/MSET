#include "hip/hip_runtime.h"
#include "utility.cuh"

// Utility routine to perform complex pointwise calculations (add, subs, mult, conj ...)

__global__ void CuPointwiseAdd(hipfftComplex *a, hipfftComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCaddf(a[i], b[i]);
    }
    return;
}

__global__ void CuPointwiseAdd(hipfftComplex a, hipfftComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCaddf(a, b[i]);
    }
    return;
}

__global__ void CuPointwiseAdd(float a, float *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a + b[i];
    }
    return;
}

__global__ void CuPointwiseAdd(float *a, float *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a[i] + b[i];
    }
    return;
}

__global__ void CuPointwiseAdd(hipfftDoubleComplex *a, hipfftDoubleComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCadd(a[i], b[i]);
    }
    return;
}

__global__ void CuPointwiseAdd(hipfftDoubleComplex a, hipfftDoubleComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCadd(a, b[i]);
    }
    return;
}

__global__ void CuPointwiseAdd(double a, double *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a + b[i];
    }
    return;
}

__global__ void CuPointwiseAdd(double *a, double *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a[i] + b[i];
    }
    return;
}


__global__ void CuPointwiseSub(hipfftComplex *a, hipfftComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCsubf(a[i], b[i]);
    }
    return;
}

__global__ void CuPointwiseSub(hipfftComplex a, hipfftComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCsubf(a, b[i]);
    }
    return;
}

__global__ void CuPointwiseSub(hipfftComplex *a, hipfftComplex b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = hipCsubf(a[i], b);
    }
    return;
}

__global__ void CuPointwiseSub(hipfftDoubleComplex *a, hipfftDoubleComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCsub(a[i], b[i]);
    }
    return;
}

__global__ void CuPointwiseSub(hipfftDoubleComplex a, hipfftDoubleComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCsub(a, b[i]);
    }
    return;
}

__global__ void CuPointwiseSub(hipfftDoubleComplex *a, hipfftDoubleComplex b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = hipCsub(a[i], b);
    }
    return;
}


__global__ void CuPointwiseSub(float *a, float *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a[i]- b[i];
    }
    return;
}

__global__ void CuPointwiseSub(float a, float *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a - b[i];
    }
    return;
}

__global__ void CuPointwiseSub(float *a, float b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = a[i] - b;
    }
    return;
}

__global__ void CuPointwiseSub(double *a, double *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a[i] - b[i];
    }
    return;
}

__global__ void CuPointwiseSub(double a, double *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a - b[i];
    }
    return;
}

__global__ void CuPointwiseSub(double *a, double b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = a[i]- b;
    }
    return;
}


__global__ void CuPointwiseMul(hipfftComplex *a, hipfftComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    hipfftComplex c;
    for (int i = threadID; i < size; i += numThreads)
    {
        c = hipCmulf(a[i], b[i]);
        b[i] = make_hipFloatComplex(hipCrealf(c), hipCimagf(c));
    }
    return;
}

__global__ void CuPointwiseMul(hipfftComplex *a, hipfftComplex *b, size_t size, size_t start_p)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    hipfftComplex c;
    for (int i = threadID; i < size; i += numThreads)
    {
        c = hipCmulf(a[i+start_p*size], b[i]);
        b[i] = make_hipFloatComplex(hipCrealf(c), hipCimagf(c));
    }
    return;
}

__global__ void CuPointwiseMul(hipfftComplex a, hipfftComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    hipfftComplex c;
    for (int i = threadID; i < size; i += numThreads)
    {
        c = hipCmulf(a, b[i]);
        b[i] = make_hipFloatComplex(hipCrealf(c), hipCimagf(c));
    }
    return;
}

__global__ void CuPointwiseMul(float a, float *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a * b[i];
    }
    return;
}

__global__ void CuPointwiseMul(hipfftDoubleComplex *a, hipfftDoubleComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCmul(a[i], b[i]);
    }
    return;
}

__global__ void CuPointwiseMul(hipfftDoubleComplex *a, hipfftDoubleComplex *b, size_t size, size_t start_p)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCmul(a[i+start_p*size], b[i]);
    }
    return;
}

__global__ void CuPointwiseMul(hipfftDoubleComplex a, hipfftDoubleComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCmul(a, b[i]);
    }
    return;
}

__global__ void CuPointwiseMul(double a, double *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;

    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a * b[i];
    }
    return;
}


__global__ void CuPointwiseDiv(hipfftComplex *a, hipfftComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCdivf(a[i], b[i]);
    }
    return;
}

__global__ void CuPointwiseDiv(float *a, float *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a[i]/b[i];
    }
    return;
}


__global__ void CuPointwiseDiv(hipfftDoubleComplex *a, hipfftDoubleComplex *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = hipCdiv(a[i], b[i]);
    }
    return;
}

__global__ void CuPointwiseDiv(double *a, double *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        b[i] = a[i]/b[i];
    }
    return;
}


__global__ void CuPointwiseExp(hipfftComplex *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    hipfftComplex res;
    float s, c, e;

    for (int i = threadID; i < size; i += numThreads)
    {
        e = expf(a[i].x);
        sincosf(a[i].y, &s, &c);
        res.x = c * e;
        res.y = s * e;
        a[i] = res;
    }
    return;
}

__global__ void CuPointwiseExp(hipfftDoubleComplex *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    hipfftDoubleComplex res;
    float s, c, e;

    for (int i = threadID; i < size; i += numThreads)
    {
        e = exp(a[i].x);
        sincos(a[i].y, &s, &c);
        res.x = c * e;
        res.y = s * e;
        a[i] = res;
    }
    return;
}

__global__ void CuPointwiseAbsSquare(float *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = a[i]*a[i];
    }
}

__global__ void CuPointwiseAbsSquare(double *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = a[i]*a[i];
    }
}


__global__ void CuPointwiseAbsSquare(hipfftComplex *a, float *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    float tmp;
    for (int i = threadID; i < size; i += numThreads)
    {
        tmp = hipCabsf(a[i]); 
        b[i] = tmp*tmp;
    }
}

__global__ void CuPointwiseAbsSquare(hipfftDoubleComplex *a, double *b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    double tmp;
    for (int i = threadID; i < size; i += numThreads)
    {
        tmp = hipCabs(a[i]); 
        b[i] = tmp*tmp;
    }
}


__global__ void CuPointwiseAbs(hipfftComplex *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = make_hipFloatComplex(hipCabsf(a[i]), 0.0f);
    }
}

__global__ void CuPointwiseAbs(hipfftDoubleComplex *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = make_hipDoubleComplex(hipCabs(a[i]), 0.0f);
    }
}


__global__ void CuPointwiseAbs(float *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = fabsf(a[i]);
    }
}

__global__ void CuPointwiseAbs(double *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = fabs(a[i]);
    }
}

__global__ void CuPointwisePow(float *a, float b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = powf(a[i], b);
    }
}

__global__ void CuPointwisePow(double *a, double b, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = pow(a[i], b);
    }
}


__global__ void CuPointwiseConj(hipfftComplex *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = hipConjf(a[i]);
    }
}

__global__ void CuPointwiseConj(hipfftDoubleComplex *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        a[i] = hipConj(a[i]);
    }
}

__global__ void CuPositivity(float *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        if (a[i] < 0){
            a[i] = 0.0f;
        }   
    }
}

__global__ void CuPositivity(double *a, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        if (a[i] < 0){
            a[i] = 0.0;
        }   
    }
}

// ---------------------------------------------------
__global__ void datatransfer_F2C(float2 *f2, float *f, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        f2[i].x =  f[i];
        f2[i].y = 0;
    }
    return;
}

__global__ void datatransfer_F2C(double2 *f2, double *f, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        f2[i].x =  f[i];
        f2[i].y = 0;
    }
    return;
}

__global__ void datatransfer_C2F(float *f, float2 *f2, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        f[i] = f2[i].x;
    }
    return;
}

__global__ void datatransfer_C2F(double *f, double2 *f2, size_t size)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    for (int i = threadID; i < size; i += numThreads)
    {
        f[i] = f2[i].x;
    }
    return;
}

__global__ void circular_shift2D_dev(float2 *f1, float2 *f2, int dims1, int dims2, int N1_shift, int N2_shift)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int n1, n2; 
    for (int i = threadID; i < dims1*dims2; i += numThreads)
    {
        n1 = (i/dims2 + N1_shift)%dims1;
        n2 = (i%dims2 + N2_shift)%dims2; 
        if (n1 < 0){
            n1 = n1 + dims1;     
        }
        if (n2 < 0){
            n2 = n2 + dims2;
        }

        f2[n1*dims2 + n2].x = f1[i].x;
        f2[n1*dims2 + n2].y = f1[i].y;
    }
    return;

}

__global__ void circular_shift2D_dev(float *f1, float *f2, int dims1, int dims2, int N1_shift, int N2_shift)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int n1, n2;
    for (int i = threadID; i < dims1*dims2; i += numThreads)
    {
        n1 = (i/dims2 + N1_shift)%dims1;
        n2 = (i%dims2 + N2_shift)%dims2; 
        if (n1 < 0){
            n1 = n1 + dims1;     
        }
        if (n2 < 0){
            n2 = n2 + dims2;
        }  
    
        f2[n1*dims2 + n2] = f1[i];
    }
    return;

}

__global__ void circular_shift2D_dev(double2 *f1, double2 *f2, int dims1, int dims2, int N1_shift, int N2_shift)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int n1, n2;
    for (int i = threadID; i < dims1*dims2; i += numThreads)
    {
        n1 = (i/dims2 + N1_shift)%dims1;
        n2 = (i%dims2 + N2_shift)%dims2; 
        if (n1 < 0){
            n1 = n1 + dims1;     
        }
        if (n2 < 0){
            n2 = n2 + dims2;
        }

        f2[n1*dims2 + n2].x = f1[i].x;
        f2[n1*dims2 + n2].y = f1[i].y;
    }
    return;

}

__global__ void circular_shift2D_dev(double *f1, double *f2, int dims1, int dims2, int N1_shift, int N2_shift)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int n1, n2;
    for (int i = threadID; i < dims1*dims2; i += numThreads)
    {
        n1 = (i/dims2 + N1_shift)%dims1;
        n2 = (i%dims2 + N2_shift)%dims2; 
        if (n1 < 0){
            n1 = n1 + dims1;     
        }
        if (n2 < 0){
            n2 = n2 + dims2;
        }  
    
        f2[n1*dims2 + n2] = f1[i];
    }
    return;

}



void circular_shift2D(float2 *f1, float2 *f2, int dims1, int dims2, int N1_shift, int N2_shift)
{
    int n1, n2;
    for (int i = 0; i < dims1*dims2; ++i){
        n1 = (i/dims2 + N1_shift)%dims1;
        n2 = (i%dims2 + N2_shift)%dims2; 
        if (n1 < 0){
            n1 = n1 + dims1;     
        }
        if (n2 < 0){
            n2 = n2 + dims2;
        }

        f2[n1*dims2 + n2].x = f1[i].x;
        f2[n1*dims2 + n2].y = f1[i].y;
    }
    return;
}

void circular_shift2D(float *f1, float *f2, int dims1, int dims2, int N1_shift, int N2_shift)
{
    int n1, n2;
    for (int i = 0; i < dims1*dims2; ++i){
        n1 = (i/dims2 + N1_shift)%dims1;
        n2 = (i%dims2 + N2_shift)%dims2; 
        if (n1 < 0){
            n1 = n1 + dims1;     
        }
        if (n2 < 0){
            n2 = n2 + dims2;
        }

        f2[n1*dims2 + n2] = f1[i];
    }
    return;
}

void circular_shift2D(double2 *f1, double2 *f2, int dims1, int dims2, int N1_shift, int N2_shift)
{
    int n1, n2;
    for (int i = 0; i < dims1*dims2; ++i){
        n1 = (i/dims2 + N1_shift)%dims1;
        n2 = (i%dims2 + N2_shift)%dims2; 
        if (n1 < 0){
            n1 = n1 + dims1;     
        }
        if (n2 < 0){
            n2 = n2 + dims2;
        }

        f2[n1*dims2 + n2].x = f1[i].x;
        f2[n1*dims2 + n2].y = f1[i].y;
    }
    return;
}

void circular_shift2D(double *f1, double *f2, int dims1, int dims2, int N1_shift, int N2_shift)
{
    int n1, n2;
    for (int i = 0; i < dims1*dims2; ++i){
        n1 = (i/dims2 + N1_shift)%dims1;
        n2 = (i%dims2 + N2_shift)%dims2; 
        if (n1 < 0){
            n1 = n1 + dims1;     
        }
        if (n2 < 0){
            n2 = n2 + dims2;
        }

        f2[n1*dims2 + n2] = f1[i];
    }
    return;
}

// Fourier shift factor
__global__ void fourier_shift_factor_dev(hipfftComplex *f, int dims1, int dims2, float N1_shift, float N2_shift)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    float n1, n2;
    hipfftComplex res;
    float s, c, e;
    
    for (int i = threadID; i < dims1*dims2; i += numThreads)
    {   
        n1 = (float) (i/dims2) - (float) dims1/2.0f;
        n2 = (float) (i%dims2) - (float) dims2/2.0f;

        f[i].x = 0;
        f[i].y = 2.0f * 3.1415926f * (N1_shift * (float) n1/ (float) dims1 + N2_shift * (float) n2/ (float) dims2);

        e = expf(f[i].x);
        sincosf(f[i].y, &s, &c);
        res.x = c * e;
        res.y = s * e;
        f[i] = res;        
    }
    return;

}

__global__ void fourier_shift_factor_dev(hipfftDoubleComplex *f, int dims1, int dims2, double N1_shift, double N2_shift)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    double n1, n2;
    hipfftDoubleComplex res;
    double s, c, e;
    
    for (int i = threadID; i < dims1*dims2; i += numThreads)
    {   
        n1 = (double) (i/dims2) - (double) dims1/2.0;
        n2 = (double) (i%dims2) - (double) dims2/2.0;

        f[i].x = 0;
        f[i].y = 2.0 * 3.1415926 * (N1_shift * (double) n1/ (double) dims1 + N2_shift * (double) n2/ (double) dims2);

        e = exp(f[i].x);
        sincos(f[i].y, &s, &c);
        res.x = c * e;
        res.y = s * e;
        f[i] = res;        
    }
    return;

}

// gradient 2D array -> 2D array
__global__ void gradient_2D_dev(hipfftComplex *f1, hipfftComplex *f2, int dims1, int dims2, int option)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int n1, n2; 

    if (option == 1){
        for (int i = threadID; i < dims1*dims2; i += numThreads)
        {
            n1 = i/dims2;
            n2 = i%dims2;
            if (n1 == 0){
                f2[i].x = f1[i+dims2].x - f1[i].x;
                f2[i].y = f1[i+dims2].y - f1[i].y;
            }            
            else if (n1 == dims1-1){
                f2[i].x = f1[i].x - f1[i-dims2].x;
                f2[i].y = f1[i].y - f1[i-dims2].y;
            }
            else{
                f2[i].x = (f1[i+dims2].x - f1[i-dims2].x)/2.0f;
                f2[i].y = (f1[i+dims2].y - f1[i-dims2].y)/2.0f;
            }
        }    

    }
    else if(option == 2){
        for (int i = threadID; i < dims1*dims2; i += numThreads)
        {
            n1 = i/dims2;
            n2 = i%dims2;
            if (n2 == 0){
                f2[i].x = f1[i+1].x - f1[i].x;
                f2[i].y = f1[i+1].y - f1[i].y;
            }            
            else if (n2 == dims2-1){
                f2[i].x = f1[i].x - f1[i-1].x;
                f2[i].y = f1[i].y - f1[i-1].y;
            }
            else{
                f2[i].x = (f1[i+1].x - f1[i-1].x)/2.0f;
                f2[i].y = (f1[i+1].y - f1[i-1].y)/2.0f;
            }
        }
    }     
    return;

}

__global__ void gradient_2D_dev(hipfftDoubleComplex *f1, hipfftDoubleComplex *f2, int dims1, int dims2, int option)
{
    const int numThreads = blockDim.x * gridDim.x;
    const int threadID = blockIdx.x * blockDim.x + threadIdx.x;
    int n1, n2; 

    if (option == 1){
        for (int i = threadID; i < dims1*dims2; i += numThreads)
        {
            n1 = i/dims2;
            n2 = i%dims2;
            if (n1 == 0){
                f2[i].x = f1[i+dims2].x - f1[i].x;
                f2[i].y = f1[i+dims2].y - f1[i].y;
            }            
            else if (n1 == dims1-1){
                f2[i].x = f1[i].x - f1[i-dims2].x;
                f2[i].y = f1[i].y - f1[i-dims2].y;
            }
            else{
                f2[i].x = (f1[i+dims2].x - f1[i-dims2].x)/2.0;
                f2[i].y = (f1[i+dims2].y - f1[i-dims2].y)/2.0;
            }
        }    

    }
    else if(option == 2){
        for (int i = threadID; i < dims1*dims2; i += numThreads)
        {
            n1 = i/dims2;
            n2 = i%dims2;
            if (n2 == 0){
                f2[i].x = f1[i+1].x - f1[i].x;
                f2[i].y = f1[i+1].y - f1[i].y;
            }            
            else if (n2 == dims2-1){
                f2[i].x = f1[i].x - f1[i-1].x;
                f2[i].y = f1[i].y - f1[i-1].y;
            }
            else{
                f2[i].x = (f1[i+1].x - f1[i-1].x)/2.0;
                f2[i].y = (f1[i+1].y - f1[i-1].y)/2.0;
            }
        }
    }     
    return;

}


float Cusum_real(hipfftComplex *f, size_t size)
{
    float output = 0.0f;
    float2 *tmp_result1;
    tmp_result1 = new float2[size]; 

    hipMemcpy(tmp_result1, f, sizeof(float2)*size, hipMemcpyDeviceToHost);
    for(size_t i=0; i<size; i++){
        output += tmp_result1[i].x; 
    }
    //std::cout << output << std::endl;

    delete [] tmp_result1; 
    return output;
}


// print array for debugging

void print_array_dev(float *a, size_t dims1, size_t dims2)
{
    
    float *tmp_result1;
    tmp_result1 = new float[dims1*dims2]; 

    hipMemcpy(tmp_result1, a, sizeof(float)*dims1*dims2, hipMemcpyDeviceToHost);

    for(size_t x=0; x<dims2; x++){
        for(size_t y=0; y<dims1; y++){
            std::cout.precision(3);
            std::cout <<"[" <<std::scientific<< tmp_result1[(y*dims2)+x] << "] ";
        }
        std::cout<<std::endl;
    }

    delete [] tmp_result1; 
}

void print_array_dev(hipfftComplex *a, size_t dims1, size_t dims2)
{
    
    float2 *tmp_result1;
    tmp_result1 = new float2[dims1*dims2]; 

    hipMemcpy(tmp_result1, a, sizeof(float2)*dims1*dims2, hipMemcpyDeviceToHost);

    for(size_t x=0; x<dims2; x++){
        for(size_t y=0; y<dims1; y++){
            std::cout.precision(3);
            std::cout <<"[" <<std::scientific<< tmp_result1[(y*dims2)+x].x << ", " << tmp_result1[(y*dims2)+x].y << "] ";
        }
        std::cout<<std::endl;
    }

    delete [] tmp_result1; 
}

void print_array_dev(double *a, size_t dims1, size_t dims2)
{
    
    double *tmp_result1;
    tmp_result1 = new double[dims1*dims2]; 

    hipMemcpy(tmp_result1, a, sizeof(double)*dims1*dims2, hipMemcpyDeviceToHost);

    for(size_t x=0; x<dims2; x++){
        for(size_t y=0; y<dims1; y++){
            std::cout.precision(3);
            std::cout <<"[" <<std::scientific<< tmp_result1[(y*dims2)+x] << "] ";
        }
        std::cout<<std::endl;
    }

    delete [] tmp_result1; 
}


void print_array_dev(hipfftDoubleComplex *a, size_t dims1, size_t dims2)
{
    
    double2 *tmp_result1;
    tmp_result1 = new double2[dims1*dims2]; 

    hipMemcpy(tmp_result1, a, sizeof(double2)*dims1*dims2, hipMemcpyDeviceToHost);

    for(size_t x=0; x<dims2; x++){
        for(size_t y=0; y<dims1; y++){
            std::cout.precision(3);
            std::cout <<"[" <<std::scientific<< tmp_result1[(y*dims2)+x].x << ", " << tmp_result1[(y*dims2)+x].y << "] ";
        }
        std::cout<<std::endl;
    }

    delete [] tmp_result1; 
}


void print_array(float *a, size_t dims1, size_t dims2)
{

     for(size_t x=0; x<dims2; x++){
        for(size_t y=0; y<dims1; y++){
            std::cout.precision(3);
            std::cout <<"[" <<std::scientific<< a[(y*dims2)+x] << "] ";
        }
        std::cout<<std::endl;
    }

}

void print_array(float2 *a, size_t dims1, size_t dims2)
{
    for(size_t x=0; x<dims2; x++){
        for(size_t y=0; y<dims1; y++){
            std::cout.precision(3);
            std::cout <<"[" <<std::scientific<< a[(y*dims2)+x].x << ", " << a[(y*dims2)+x].y << "] ";
        }
        std::cout<<std::endl;
    }

}

void print_array(double *a, size_t dims1, size_t dims2)
{
        for(size_t x=0; x<dims2; x++){
        for(size_t y=0; y<dims1; y++){
            std::cout.precision(3);
            std::cout <<"[" <<std::scientific<< a[(y*dims2)+x] << "] ";
        }
        std::cout<<std::endl;
    }

}


void print_array(double2 *a, size_t dims1, size_t dims2)
{
        for(size_t x=0; x<dims2; x++){
        for(size_t y=0; y<dims1; y++){
            std::cout.precision(3);
            std::cout <<"[" <<std::scientific<< a[(y*dims2)+x].x << ", " << a[(y*dims2)+x].y << "] ";
        }
        std::cout<<std::endl;
    }

}




